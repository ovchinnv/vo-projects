#ifdef __TEX
 texture<__CTYPE> tex_reduce; // may not work for types other than float
#endif

#define _BDIM 128 // threadblock size
#define _VPT 32    // values per thread

#include <stdio.h>
#include "kernel.cu"

#define _NBLK(N,n) (N)/(n) + ( (N) % (n) > 0 )


extern "C" void reduction_c(__CTYPE *A, int n, __CTYPE *val) {


  __CTYPE *devA, *devAout ;
  unsigned int numblk = _NBLK ( n , 2*_BDIM*_VPT ) ;
  __CTYPE redA[numblk];

// if (n<10000) 
//for (int i=0;i<n;i++) { printf("%12.5f\n",A[i]);}

 printf("N :%5d\n",n);

  hipMalloc( &devA, n*sizeof(__CTYPE) ); // device copy of A
  hipMemcpy( devA, A, n*sizeof(__CTYPE), hipMemcpyHostToDevice);

  hipMalloc( &devAout, numblk*sizeof(__CTYPE) ); // device copy of reduced array data

  dim3 block ( _BDIM, 1, 1 ); // thread indices
  dim3 grid  ( numblk, 1, 1 ); // block indices
 printf("threads/block :%5d\n",_BDIM);
 printf("blocks :%5d\n",numblk);
 printf("values/thread :%5d\n",_VPT);

// launch kernel

#ifdef __TEX
  hipBindTexture(NULL, tex_reduce, devA, n*sizeof(__CTYPE));
#endif
 for (int i=0 ; i<1000;i++){
  reduction_cuda<<<grid,block>>>(devA, devAout, n);
//  reduction_cuda_simple<<<grid,block>>>(devA, devAout, n);
// reduction_cuda_simple2<<<grid,block>>>(devA, devAout, n);
// reduction_cuda_simple3<<<grid,block>>>(devA, devAout, n);
 }
#ifdef __TEX
  hipUnbindTexture(tex_reduce);
#endif

// copy partially reduced array
  hipMemcpy( redA, devAout, numblk*sizeof(__CTYPE), hipMemcpyDeviceToHost);
//
// compute final value
//
#define _FMT "%12.5f\n"
#define _FMT "%5d\n"
  *val=0.f ; for (int i=0;i<numblk;i++) { *val+=redA[i] ;}//  printf(_FMT,redA[i]);}
  printf(_FMT,*val);
// free device memory
  hipFree(devA);
  hipFree(devAout);

}