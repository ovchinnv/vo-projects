
#include <hip/hip_runtime_api.h>

extern "C" void AllocDevMem(__CUFLOAT **p, __CINT n) {
// note that p is a double pointer, so that we do not have to take its address
// not sure everything is done correctly
// hipMalloc(&p, n*sizeof(__CUFLOAT));
 checkCudaErrors(hipMalloc(p, n*sizeof(__CUFLOAT)));
}

extern "C" void FreeDevMem(__CUFLOAT **p) {
// hipFree(p);
 checkCudaErrors(hipFree(*p));
}

extern "C" void CopyHostToDevice(__CFLOAT *hostp, __CUFLOAT *devp, __CINT n){
 checkCudaErrors(hipMemcpy(devp, hostp, sizeof(__CUFLOAT)*n, hipMemcpyHostToDevice));
}

extern "C" void CopyDeviceToHost(__CFLOAT *hostp, __CUFLOAT *devp, __CINT n){
 checkCudaErrors(hipMemcpy(hostp, devp, sizeof(__CUFLOAT)*n, hipMemcpyDeviceToHost));
}
