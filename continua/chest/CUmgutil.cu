#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include "mgkernels.cu"

extern "C" void AllocDevMem(__CUFLOAT **p, __CINT n) {
// note that p is a double pointer, so that we do not have to take its address
// not sure everything is done correctly
// hipMalloc(&p, n*sizeof(__CUFLOAT));
 checkCudaErrors(hipMalloc(p, n*sizeof(__CUFLOAT)));
}

extern "C" void FreeDevMem(__CUFLOAT **p) {
// hipFree(p);
 checkCudaErrors(hipFree(*p));
}

extern "C" void CopyHostToDevice(__CFLOAT *hostp, __CUFLOAT *devp, __CINT n){
 checkCudaErrors(hipMemcpy(devp, hostp, sizeof(__CUFLOAT)*n, hipMemcpyHostToDevice));
}

extern "C" void CopyDeviceToHost(__CFLOAT *hostp, __CUFLOAT *devp, __CINT n){
 checkCudaErrors(hipMemcpy(hostp, devp, sizeof(__CUFLOAT)*n, hipMemcpyDeviceToHost));
}

extern "C" void GaussSeidel_Cuda(__CUFLOAT *devp, __CUFLOAT *devrhs, __CUFLOAT *deveps, __CUFLOAT *devkappa, __CUFLOAT *devdx, __CUFLOAT *devdy, __CUFLOAT *devdz,
                                 __CINT i3b, __CINT i3, __CINT i1, __CINT j1, __CINT k1, __CINT nx, __CINT ny, __CINT nz, __CFLOAT dt, int8_t i2d) {

 int nnx=nx-2; // inner points
 int nny=ny-2;
//
 if (!i2d) {
  dim3 block(_BSIZE_X, _BSIZE_Y);
  dim3 grid( nnx / (_SX*_BSIZE_X) + ( nnx % (_SX*_BSIZE_X) > 0 ), nny / (_SY*_BSIZE_Y) + ( nnx % (_SY*_BSIZE_Y) > 0 ));
//
  Gauss_Seidel_Cuda_3D<<<grid, block>>>(devp, devrhs, deveps, devkappa, devdx, devdy, devdz, i3b, i3, i1, j1, k1, nx, ny, nz, dt);
 }
}

extern "C" void ApplyBC_Cuda(__CUFLOAT *devp, __CUFLOAT *devbcw, __CUFLOAT *devbce, __CUFLOAT *devbcn, __CUFLOAT *devbcs, __CUFLOAT *devbcf, __CUFLOAT *devbcb,
                                 __CINT i3b, __CINT i2, __CINT j2, __CINT k2, __CINT nx, __CINT ny, __CINT nz, __CINT *bc_type, __CFLOAT *bc_wgt, int8_t i2d) {

}
